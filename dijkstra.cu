
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define INF INT_MAX
#define NUMBEROFVERTICES 501
#define NUMBEROFEDGES 998

int Ma_notEmpty(int *);

int Ma_notEmpty(int *Ma)
{
	int i, sum = 0;
	for(i=0; i <  NUMBEROFVERTICES; i++)
	{
		sum += Ma[i];
	}
	return sum;
}




__global__ void CUDA_SSSP_KERNEL1(int *Va, int *Ea, int *Wa, int *Ca, int *Ma, int *Ua)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int nid, i;


	if(Ma[tid])
	{
		Ma[tid] = 0;
		for (i = Va[tid]; i < Va[tid+1]; i++)
		{
			nid = Ea[i];
			if (Ua[nid] > Ca[tid] + Wa[nid])
			{
				Ua[nid] = Ca[tid] + Wa[nid];
			}
		}
	}
	
}

__global__ void CUDA_SSSP_KERNEL2(int *Va, int *Ea, int *Wa, int *Ca, int *Ma, int *Ua)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(Ca[tid] > Ua[tid])
	{
		Ca[tid] = Ua[tid];
		Ma[tid] = 1;
	}
	Ua[tid] = Ca[tid];
}

int main(int argc, char **argv)
{
	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);
	float millisecond = 0;
	int adjacency_matrix[NUMBEROFVERTICES][NUMBEROFVERTICES];
	int  j;
	int sum = 0, e = 0;
	
	int S = 0, i;

	int h_Va[NUMBEROFVERTICES];
	int h_Ea[NUMBEROFEDGES];
	int h_Wa[NUMBEROFEDGES];
	
	int h_Ca[NUMBEROFVERTICES];
	int h_Ma[NUMBEROFVERTICES];
	int h_Ua[NUMBEROFVERTICES];

	int V_BYTESIZE = NUMBEROFVERTICES * sizeof(int);
	int E_BYTESIZE = NUMBEROFEDGES * sizeof(int);

	int *d_Va, *d_Ea, *d_Wa, *d_Ca, *d_Ma, *d_Ua;
	srand(time(NULL));

	for (i=0; i < NUMBEROFVERTICES; i++)
	{
		h_Ca[i] = INF;
		h_Ma[i] = 0;
		h_Ua[i] = INF;
	}

//==================================================================
/*memset(adjacency_matrix, 0, sizeof(int) * NUMBEROFVERTICES * NUMBEROFVERTICES);
for (i = 0; i < NUMBEROFVERTICES -1; i++)
{
	for (j = 0; j < NUMBEROFVERTICES -1; j++)
	{
		if (j == (i + 1) || i == (j + 1))
			adjacency_matrix[i][j] = 1;
		else
			adjacency_matrix[i][j] = 0;
	}

}

adjacency_matrix[NUMBEROFVERTICES -2][NUMBEROFVERTICES -2] = 1;*/
//==================================================================
	memset(adjacency_matrix, 0, sizeof(int) * NUMBEROFVERTICES * NUMBEROFVERTICES);
	for(i = 0; i < NUMBEROFVERTICES -1; i++)
	{
		for(j = 0; j < NUMBEROFVERTICES -1; j++)
		{
			if(j == (2 * i + 1) || i == (2 * j + 1))
				adjacency_matrix[i][j] = 1;
			else if(j == (2 * i + 2) || i == (2 * j + 2))
				adjacency_matrix[i][j] = 1;
		}
	}
 

	

	for (i = 0; i < NUMBEROFVERTICES-1; i++)
	{
		h_Va[i] = sum;
		for (j = 0; j < NUMBEROFVERTICES-1; j++)
		{
			if (adjacency_matrix[i][j] == 1)
			{
				h_Ea[e++] = j;
			}
			sum += adjacency_matrix[i][j];	 
		}
	}

	for (i=0; i < NUMBEROFEDGES; i++)
	{
		//h_Wa[i] = 1;
		h_Wa[i] = (rand() % 10) + 1;
	}

	/*for (i = 0; i < NUMBEROFVERTICES; i++)
	{
		printf("V[%d]= %d \t", i, h_Va[i]);
	}
	printf("\n");
	
	for (i = 0; i < NUMBEROFEDGES; i++)
	{
		printf("E[%d]= %d \tW[%d]= %d", i, h_Ea[i], i, h_Wa[i]);
	}*/
//==================================================================	
//	createTree(h_Va, h_Ea);	
	/*h_Va[0] = 0;
	h_Va[1] = 3;
	h_Va[2] = 5;
	h_Va[3] = 7;
	h_Va[4] = 8;
	h_Va[5] = 10;
	h_Va[6] = 12;
	

	h_Ea[0] = 1;
	h_Ea[1] = 2;
	h_Ea[2] = 3;
	h_Ea[3] = 0;
	h_Ea[4] = 5;
	h_Ea[5] = 0;
	h_Ea[6] = 4;
	h_Ea[7] = 0;
	h_Ea[8] = 2;
	h_Ea[9] = 5;
	h_Ea[10] = 1;
	h_Ea[11] = 4;
	

	h_Wa[0] = 1;
	h_Wa[1] = 1;
	h_Wa[2] = 1;
	h_Wa[3] = 1;
	h_Wa[4] = 1;
	h_Wa[5] = 1;
	h_Wa[6] = 1;
	h_Wa[7] = 1;
	h_Wa[8] = 1;
	h_Wa[9] = 1;
	h_Wa[10] = 1;
	h_Wa[11] = 1;*/
	
	
	hipMalloc((void **) &d_Va, V_BYTESIZE);
	hipMalloc((void **) &d_Ea, E_BYTESIZE);
	hipMalloc((void **) &d_Wa, E_BYTESIZE);
	hipMalloc((void **) &d_Ca, V_BYTESIZE);
	hipMalloc((void **) &d_Ma, V_BYTESIZE);
	hipMalloc((void **) &d_Ua, V_BYTESIZE);

	hipMemcpy(d_Va, h_Va, V_BYTESIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_Ea, h_Ea, E_BYTESIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_Wa, h_Wa, E_BYTESIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_Ma, h_Ma, V_BYTESIZE, hipMemcpyHostToDevice);
	h_Ma[S] = 1;
	h_Ca[S] = 0;
	h_Ua[S] = 0;
	
	hipEventRecord(begin);
	while(Ma_notEmpty(h_Ma))
	{
		hipMemcpy(d_Ca, h_Ca, V_BYTESIZE, hipMemcpyHostToDevice);
		hipMemcpy(d_Ua, h_Ua, V_BYTESIZE, hipMemcpyHostToDevice);
		hipMemcpy(d_Ma, h_Ma, V_BYTESIZE, hipMemcpyHostToDevice);

		CUDA_SSSP_KERNEL1<<<1, (NUMBEROFVERTICES-1)>>>(d_Va, d_Ea, d_Wa, d_Ca, d_Ma, d_Ua);

		hipMemcpy(h_Ca, d_Ca, V_BYTESIZE, hipMemcpyDeviceToHost);
		hipMemcpy(h_Ua, d_Ua, V_BYTESIZE, hipMemcpyDeviceToHost);
		hipMemcpy(h_Ma, d_Ma, V_BYTESIZE, hipMemcpyDeviceToHost);
		hipMemcpy(d_Ca, h_Ca, V_BYTESIZE, hipMemcpyHostToDevice);
		hipMemcpy(d_Ua, h_Ua, V_BYTESIZE, hipMemcpyHostToDevice);
		hipMemcpy(d_Ma, h_Ma, V_BYTESIZE, hipMemcpyHostToDevice);		


		CUDA_SSSP_KERNEL2<<<1, (NUMBEROFVERTICES-1)>>>(d_Va, d_Ea, d_Wa, d_Ca, d_Ma, d_Ua);
		//printf("_____________________________\n");

		hipMemcpy(h_Ma, d_Ma, V_BYTESIZE, hipMemcpyDeviceToHost);
		hipMemcpy(h_Ca, d_Ca, V_BYTESIZE, hipMemcpyDeviceToHost);
		hipMemcpy(h_Ua, d_Ua, V_BYTESIZE, hipMemcpyDeviceToHost);
		hipMemcpy(d_Ca, h_Ca, V_BYTESIZE, hipMemcpyHostToDevice);
		hipMemcpy(d_Ma, h_Ma, V_BYTESIZE, hipMemcpyHostToDevice);
		hipMemcpy(d_Ua, h_Ua, V_BYTESIZE, hipMemcpyHostToDevice);

		/*for(i=0; i < NUMBEROFVERTICES; i++)
		{
			printf("Ma[%d]  Ua[%d] Ca[%d]\n",h_Ma[i], h_Ua[i], h_Ca[i]);
		}
		printf("_____________________________\n");*/
//		break;
	}

	hipMemcpy(d_Ca, h_Ca, V_BYTESIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_Ma, h_Ma, V_BYTESIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_Ua, h_Ua, V_BYTESIZE, hipMemcpyHostToDevice);
	/* print result */
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&millisecond, begin, end);
	for(i=0; i < NUMBEROFVERTICES - 1; i++)
	{
		printf("[%d]\t", h_Ca[i]);
	}

	printf("\n >> mSecond= %f\n", millisecond);

	hipFree(d_Ca);
	hipFree(d_Ma);
	hipFree(d_Ua);
	hipFree(d_Va);
	hipFree(d_Ea);
	hipFree(d_Wa);

	return EXIT_SUCCESS;
}
